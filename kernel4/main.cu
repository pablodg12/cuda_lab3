
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void kernelRed(int *A, int *x, int *b, int N){
  extern __shared__ int sm[];
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  if(tId < N){
    for(int k=0; k < N; k++){
      sm[threadIdx.x] = A[(int)(k*1e4+tId)]*x[tId];
      __syncthreads();
      if(threadIdx.x < 128){sm[threadIdx.x] += sm[threadIdx.x+128];__syncthreads();}
      if(threadIdx.x < 64){sm[threadIdx.x] += sm[threadIdx.x+64];__syncthreads();}
      if(threadIdx.x < 32){sm[threadIdx.x] += sm[threadIdx.x+32];__syncthreads();}
      if(threadIdx.x < 16){sm[threadIdx.x] += sm[threadIdx.x+16];__syncthreads();}
      if(threadIdx.x < 8){sm[threadIdx.x] += sm[threadIdx.x+8];__syncthreads();}
      if(threadIdx.x < 4){sm[threadIdx.x] += sm[threadIdx.x+4];__syncthreads();}
      if(threadIdx.x < 2){sm[threadIdx.x] += sm[threadIdx.x+2];__syncthreads();}
      if(threadIdx.x < 1){sm[threadIdx.x] += sm[threadIdx.x+1];__syncthreads();}
      if(threadIdx.x < 1){atomicAdd(&b[k],sm[threadIdx.x]);}
    }
  }
}

int main(int argc, char const *argv[])
{
  int n = 1e4;
  int block_size = 256;
  int grid_size = (int) ceil((float) n/ block_size);

  int *GPU_b;
  int *GPU_x;
  int *GPU_A;

  int *CPU_x = (int *) malloc(1e4 * sizeof (int));
  int *CPU_A = (int *) malloc(1e8 * sizeof (int));

  for(int k = 0; k < 1e8; k++){
    if(k < 1e4){
      CPU_x[k] = 1;
    }
    CPU_A[k] = 1;
  }  

  hipMalloc(&GPU_x , 1e4 * sizeof(int));
  hipMalloc(&GPU_b , 1e4 * sizeof(int));
  hipMalloc(&GPU_A , 1e8 * sizeof(int));

  hipMemcpy(GPU_A, CPU_A, 1e8 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(GPU_x, CPU_x, 1e4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemset(GPU_b, 0, 1e4 * sizeof(int));

  kernelRed<<<grid_size, block_size, block_size*sizeof(int)>>>(GPU_A, GPU_x, GPU_b, n);

  hipMemcpy(CPU_x, GPU_b, 1e4 * sizeof(int), hipMemcpyDeviceToHost);

  for(int k = 0; k< 1e4; k++){
    printf("%d\n", CPU_x[k]);
  }

  hipFree(GPU_x);
  hipFree(GPU_b);
  hipFree(GPU_A);
  free(CPU_x);
  free(CPU_A);

  return(0);
}