
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void kernelRed(int *A, int *x, int *b, int N){
  extern __shared__ int sm[];
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  if(tId < N){
    for(int k=0; k < N; k++){
      sm[threadIdx.x] = A[(int)(k*1e4+tId)]*x[tId];
      __syncthreads();
      if(threadIdx.x < 128){sm[threadIdx.x] += sm[threadIdx.x+128];__syncthreads();}
      //if(tId<128){sm[mp] += sm[tId+128];__syncthreads();}
      //if(tId<64){sm[mp] += sm[tId+64];__syncthreads();}
      //if(tId<32){sm[mp] += sm[tId+32];__syncthreads();}
      //if(tId<16){sm[mp] += sm[tId+16];__syncthreads();}
      //if(tId<8){sm[mp] += sm[tId+8];__syncthreads();}
      //if(tId<4){sm[mp] += sm[tId+4];__syncthreads();}
      //if(tId<2){sm[mp] += sm[tId+2];__syncthreads();}
      //if(tId<1){b[tId] = sm[tId];__syncthreads();}
    }
  }
}

int main(int argc, char const *argv[])
{
  int n = 1e4;
  int block_size = 256;
  int grid_size = (int) ceil((float) n/ block_size);

  int *GPU_b;
  int *GPU_x;
  int *GPU_A;

  int *CPU_x = (int *) malloc(1e4 * sizeof (int));
  int *CPU_A = (int *) malloc(1e8 * sizeof (int));

  for(int k = 0; k < 1e8; k++){
    if(k < 1e4){
      CPU_x[k] = 1;
    }
    CPU_A[k] = 1;
  }  

  hipMalloc(&GPU_x , 1e4 * sizeof(int));
  hipMalloc(&GPU_b , 1e4 * sizeof(int));
  hipMalloc(&GPU_A , 1e8 * sizeof(int));

  hipMemcpy(GPU_A, CPU_A, 1e8 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(GPU_x, CPU_x, 1e4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemset(GPU_b, 0, 1e4 * sizeof(int));

  kernelRed<<<grid_size, block_size, block_size*sizeof(int)>>>(GPU_A, GPU_x, GPU_b, n);

  hipMemcpy(CPU_x, GPU_b, 1e4 * sizeof(int), hipMemcpyDeviceToHost);

  for(int k = 0; k< 1e4; k++){
    printf("%d\n", CPU_x[k]);
  }

  hipFree(GPU_x);
  hipFree(GPU_b);
  hipFree(GPU_A);
  free(CPU_x);
  free(CPU_A);

  return(0);
}