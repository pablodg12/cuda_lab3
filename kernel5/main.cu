
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void kernelSM(int *A, int *x, int *b, int N){
  extern __shared__ int sm[];
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  if(tId < N){
    for(int k = 0; k < N/blockDim.x; k++){
      sm[threadIdx.x] = x[threadIdx.x + 256*k];
      __syncthreads();
      b[tId] += A[(int)(tId*N+(threadIdx.x+256*k))]*sm[threadIdx.x];
      __syncthreads();
      }
    }
  }

int main(int argc, char const *argv[])
{
  int n = 1e4;
  int block_size = 256;
  int grid_size = (int) ceil((float) n/ block_size);

  int *GPU_b;
  int *GPU_x;
  int *GPU_A;

  int *CPU_x = (int *) malloc(1e4 * sizeof (int));
  int *CPU_A = (int *) malloc(1e8 * sizeof (int));

  for(int k = 0; k < 1e8; k++){
    if(k < 1e4){
      CPU_x[k] = 1;
    }
    CPU_A[k] = 1;
  }  

  hipMalloc(&GPU_x , 1e4 * sizeof(int));
  hipMalloc(&GPU_b , 1e4 * sizeof(int));
  hipMalloc(&GPU_A , 1e8 * sizeof(int));

  hipMemcpy(GPU_A, CPU_A, 1e8 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(GPU_x, CPU_x, 1e4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemset(GPU_b, 0, 1e4 * sizeof(int));

  kernelSM<<<grid_size, block_size, block_size*sizeof(int)>>>(GPU_A, GPU_x, GPU_b, n);

  hipMemcpy(CPU_x, GPU_b, 1e4 * sizeof(int), hipMemcpyDeviceToHost);

  for(int k = 0; k< 1e4; k++){
    printf("%d\n", CPU_x[k]);
  }

  hipFree(GPU_x);
  hipFree(GPU_b);
  hipFree(GPU_A);
  free(CPU_x);
  free(CPU_A);

  return(0);
}