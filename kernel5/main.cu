
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void kernelSM(int *A, int *x, int *b, int N){
  extern __shared__ int sm[];
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  int b_local = 0;
  for(int k = 0; k < N/blockDim.x+1; k++){
    int v_max = (k == (int) N/blockDim.x? 16:256);
    if(threadIdx.x < v_max){
      sm[threadIdx.x] = x[threadIdx.x + 256*k];
    }
    __syncthreads();
    if(tId < N){      
      for(int t = 0; t < v_max; t++){
        b_local += A[(int)(tId*N+(t+256*k))]*sm[t];
      }
    }
  }
  if(tId>N){
    b[tId] += b_local;
  }
}

int main(int argc, char const *argv[])
{
  int n = 1e4;
  int block_size = 256;
  int grid_size = (int) ceil((float) n/ block_size);

  int *GPU_b;
  int *GPU_x;
  int *GPU_A;

  int *CPU_x = (int *) malloc(1e4 * sizeof (int));
  int *CPU_A = (int *) malloc(1e8 * sizeof (int));

  for(int k = 0; k < 1e8; k++){
    if(k < 1e4){
      CPU_x[k] = 1;
    }
    CPU_A[k] = 1;
  }  

  hipMalloc(&GPU_x , 1e4 * sizeof(int));
  hipMalloc(&GPU_b , 1e4 * sizeof(int));
  hipMalloc(&GPU_A , 1e8 * sizeof(int));

  hipMemcpy(GPU_A, CPU_A, 1e8 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(GPU_x, CPU_x, 1e4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemset(GPU_b, 0, 1e4 * sizeof(int));

  kernelSM<<<grid_size, block_size, block_size*sizeof(int)>>>(GPU_A, GPU_x, GPU_b, n);

  hipMemcpy(CPU_x, GPU_b, 1e4 * sizeof(int), hipMemcpyDeviceToHost);

  //for(int k = 0; k< 1e4; k++){
  //  printf("%d\n", CPU_x[k]);
  //  }

  hipFree(GPU_x);
  hipFree(GPU_b);
  hipFree(GPU_A);
  free(CPU_x);
  free(CPU_A);

  return(0);
}