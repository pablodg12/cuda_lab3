
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void kernelSM(int *A, int *x, int *b, int N){
  extern __shared__ int sm[];
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  if(tId < N){
    for(int k = 0; k < N/blockDim.x + 1; k++){
      sm[threadIdx.x] = x[threadIdx.x + 256*k];
      __syncthreads();
      //int v_max = (k == (int) N/blockDim.x? 16:256);
      //for(int t = 0; t < v_max; t++){
      //  b[tId] += A[(int)(tId*N+(t+256*k))]*sm[t];
      //}
    }
  }
}

int main(int argc, char const *argv[])
{
  int n = 1e4;
  int block_size = 256;
  int grid_size = (int) ceil((float) n/ block_size);

  int *GPU_b;
  int *GPU_x;
  int *GPU_A;

  int *CPU_x = (int *) malloc(1e4 * sizeof (int));
  int *CPU_A = (int *) malloc(1e8 * sizeof (int));

  for(int k = 0; k < 1e8; k++){
    if(k < 1e4){
      CPU_x[k] = 1;
    }
    CPU_A[k] = 1;
  }  

  hipMalloc(&GPU_x , 1e4 * sizeof(int));
  hipMalloc(&GPU_b , 1e4 * sizeof(int));
  hipMalloc(&GPU_A , 1e8 * sizeof(int));

  hipMemcpy(GPU_A, CPU_A, 1e8 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(GPU_x, CPU_x, 1e4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemset(GPU_b, 0, 1e4 * sizeof(int));

  kernelSM<<<grid_size, block_size, block_size*sizeof(int)>>>(GPU_A, GPU_x, GPU_b, n);

  hipMemcpy(CPU_x, GPU_b, 1e4 * sizeof(int), hipMemcpyDeviceToHost);

  for(int k = 0; k< 1e4; k++){
    printf("%d\n", CPU_x[k]);
  }

  hipFree(GPU_x);
  hipFree(GPU_b);
  hipFree(GPU_A);
  free(CPU_x);
  free(CPU_A);

  return(0);
}