
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__constant__ int buff[10000];

__global__ void kernelCM(int *A, int *b, int N){
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  if(tId< N){
    for(int k=0; k < N; k++){
      b[tId] += A[(int)(tId*N+k)]*buff[k];
    }
  }
}

int main(int argc, char const *argv[])
{
  int n = 1e4;
  int block_size = 256;
  int grid_size = (int) ceil((float) n/ block_size);

  int *GPU_b;
  int *GPU_x;
  int *GPU_A;

  int *CPU_x = (int *) malloc(1e4 * sizeof (int));
  int *CPU_A = (int *) malloc(1e8 * sizeof (int));

  for(int k = 0; k < 1e8; k++){
    if(k < 1e4){
      CPU_x[k] = 1;
    }
    CPU_A[k] = 1;
  }  

  hipMalloc(&GPU_x , 1e4 * sizeof(int));
  hipMalloc(&GPU_b , 1e4 * sizeof(int));
  hipMalloc(&GPU_A , 1e8 * sizeof(int));

  hipMemcpy(GPU_A, CPU_A, 1e8 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(buff), CPU_x, 1e4 * sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemset(GPU_b, 0, 1e4 * sizeof(int));

  kernelCM<<<grid_size, block_size, block_size*sizeof(int)>>>(GPU_A, GPU_b, n);

  hipMemcpy(CPU_x, GPU_b, 1e4 * sizeof(int), hipMemcpyDeviceToHost);

  //for(int k = 0; k< 1e4; k++){
  //  printf("%d\n", CPU_x[k]);
  //}

  hipFree(GPU_x);
  hipFree(GPU_b);
  hipFree(GPU_A);
  free(CPU_x);
  free(CPU_A);

  return(0);
}