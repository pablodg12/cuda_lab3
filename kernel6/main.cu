
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__constant__ int buff[10000];

__global__ void kernelCM(int *A, int *b, int N){
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  if(tId< N){
    for(int k=0; k < N; k++){
      b[tId] += A[(int)(tId*N+k)]*buff[k];
    }
  }
}

int main(int argc, char const *argv[])
{
  int n = 1e4;
  int block_size = 256;
  int grid_size = (int) ceil((float) n/ block_size);

  int *GPU_b;
  int *GPU_x;
  int *GPU_A;

  int *CPU_x = (int *) malloc(n * sizeof (int));
  int *CPU_A = (int *) malloc(2*n * sizeof (int));

  for(int k = 0; k < 2*n; k++){
    if(k < n){
      CPU_x[k] = 1;
    }
    CPU_A[k] = 1;
  }  

  hipMalloc(&GPU_x , n * sizeof(int));
  hipMalloc(&GPU_b , n * sizeof(int));
  hipMalloc(&GPU_A , 2*n * sizeof(int));

  hipMemcpy(GPU_A, CPU_A, 2*n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(buff), CPU_x, n * sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemset(GPU_b, 0, n * sizeof(int));

  kernelCM<<<grid_size, block_size, block_size*sizeof(int)>>>(GPU_A, GPU_b, n);

  hipMemcpy(CPU_x, GPU_b, n * sizeof(int), hipMemcpyDeviceToHost);

  //for(int k = 0; k< n; k++){
  //  printf("%d\n", CPU_x[k]);
  //}

  hipFree(GPU_x);
  hipFree(GPU_b);
  hipFree(GPU_A);
  free(CPU_x);
  free(CPU_A);

  return(0);
}