
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void kernelA(int *A, int *x, int *b, int N){
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  for(int k=0;k<1e4;k++){
    atomicAdd(&b[k],A[(int)(k*1e4+tId)]*x[tId]);
  }
} 

int main(int argc, char const *argv[])
{
  int n = 1e4;
  int block_size = 256;
  int grid_size = (int) ceil((float) n/ block_size);

  int *GPU_b;
  int *GPU_x;
  int *GPU_A;

  int *CPU_x = (int *) malloc(1e4 * sizeof (int));
  int *CPU_A = (int *) malloc(1e8 * sizeof (int));

  for(int k = 0; k < n; k++){
    if(k < 1e4){
      CPU_x[k] = 1;
    }
    CPU_A[k] = 1;
  }  

  hipMalloc(&GPU_x , 1e4 * sizeof(int));
  hipMalloc(&GPU_b , 1e4 * sizeof(int));
  hipMalloc(&GPU_A , 1e8 * sizeof(int));

  hipMemcpy(GPU_A, CPU_A, 1e8 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(GPU_x, CPU_x, 1e4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemset(GPU_b,0,1e4 * sizeof(int));

  kernelA<<<grid_size, block_size>>>(GPU_A, GPU_x, GPU_b, n);

  hipMemcpy(CPU_x, GPU_b, 1e4 * sizeof(int), hipMemcpyDeviceToHost);

  printf("%d\n", CPU_x[0]);

  hipFree(GPU_x);
  hipFree(GPU_b);
  hipFree(GPU_A);
  free(CPU_x);
  free(CPU_A);

  return(0);
}
