
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>


__global__ void kernelA(int *A, int *x, int *b, int N){
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  int mp = tId - 1e4*floor(tId/1e4);
  A[tId] = A[tId]*x[mp];
  if(tId < 2){
    printf("%d\n", A[tId]);
  }
  atomicAdd(&b[mp],A[tId]);
} 

int main(int argc, char const *argv[])
{
  int n = 1e8;
  int block_size = 256;
  int grid_size = (int) ceil((float) n/ block_size);

  int *GPU_b;
  int *GPU_x;
  int *GPU_A;

  int *CPU_x = (int *) malloc(n*0.5 * sizeof (int));
  int *CPU_A = (int *) malloc(n * sizeof (int));

  for(int k = 0; k < n; k++){
    if(k < n){
      CPU_x[k] = 1;
    }
    CPU_A[k] = 1;
  }  

  hipMalloc(&GPU_x , n*0.5 * sizeof(int));
  hipMalloc(&GPU_b , n*0.5 * sizeof(int));
  hipMalloc(&GPU_A , n * sizeof(int));

  hipMemcpy(GPU_A, CPU_A, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(GPU_x, CPU_x, n*0.5 * sizeof(int), hipMemcpyHostToDevice);
  hipMemset(GPU_b,0,n*0.5 * sizeof(int));

  kernelA<<<grid_size, block_size>>>(GPU_A, GPU_x, GPU_b, n);

  hipMemcpy(CPU_x, GPU_b, n*0.5 * sizeof(int), hipMemcpyDeviceToHost);

  printf("%d\n", CPU_x[0]);

  hipFree(GPU_x);
  hipFree(GPU_b);
  hipFree(GPU_A);
  free(CPU_x);
  free(CPU_x);

  return(0);
}
