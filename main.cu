
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>



int main(int argc, char const *argv[])
{
  int *GPU_x;
  int *GPU_b;
  int *GPU_A;
  hipMalloc(&GPU_x , 1e4* sizeof(int));
  hipMalloc(&GPU_b , 1e4* sizeof(int));
  hipMalloc(&GPU_A , 1e8* sizeof(int));
  hipMemset(GPU_x, 1, 1e4*sizeof(int));
  hipMemset(GPU_A, 1, 1e8*sizeof(int));


  hipFree(GPU_x);
  hipFree(GPU_b);
  hipFree(GPU_A);



  return(0);
}
