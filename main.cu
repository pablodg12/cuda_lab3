
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>


__global__ void kernelA(int *A, int *x, int *b, int N){
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  int mp = tId - 1e4*floor(tId/1e4);
  A[tId] = A[tId]*x[mp];
  atomicAdd(&b[mp],A[tId]);
} 

int main(int argc, char const *argv[])
{
  int *GPU_b;
  int *GPU_x;
  int *GPU_A;

  

  int n = 1e8;
  int block_size = 256;
  int grid_size = (int) ceil((float) n/ block_size);

  int *CPU_b = (int *) malloc(n*0.5 * sizeof (int));  

  hipMalloc(&GPU_x , n*0.5 * sizeof(int));
  hipMalloc(&GPU_b , n*0.5 * sizeof(int));
  hipMalloc(&GPU_A , n * sizeof(int));

  hipMemset(GPU_x, 1, 1e4 * sizeof(int)); 
  hipMemset(GPU_b, 0, 1e4 * sizeof(int));
  hipMemset(GPU_A, 1, 1e8 * sizeof(int));

  kernelA<<<grid_size, block_size>>>(GPU_A, GPU_x, GPU_b, n);

  hipMemcpy(CPU_b, GPU_b, 1e4 * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(GPU_x);
  hipFree(GPU_b);
  hipFree(GPU_A);

  printf("%d\n", CPU_b[0]);

  return(0);
}
